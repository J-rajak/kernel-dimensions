
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <errno.h>
#include <hip/hip_runtime_api.h>

/****************************************************************************
 * An experiment with cuda kernel invocation parameters. Two threads on  
 * three blocks should yield six kernel invocations.
 *
 * Compile with:
 *   nvcc -o 02 02.cu
 * 
 * Dr Kevan Buckley, University of Wolverhampton, January 2018
 *****************************************************************************/

__global__ void kernel(){
  int i = (blockIdx.x * blockDim.x) + threadIdx.x;

  printf("gd(%4d,%4d,%4d) bd(%4d,%4d,%4d) bi(%4d,%4d,%4d) ti(%4d,%4d,%4d) %d\n",
    gridDim.x, gridDim.y, gridDim.z, 
    blockDim.x, blockDim.y, blockDim.z,
    blockIdx.x, blockIdx.y, blockIdx.z,
    threadIdx.x, threadIdx.y, threadIdx.z, i); 
}

void advice(){
  printf("\ngd = gridDim\n");
  printf("bd = blockDim\n");  
  printf("bi = blockIdx\n");  
  printf("ti = threadIdx\n\n");
}

int main() {
  hipError_t error;

  advice();

  kernel <<<2, 3>>>();
  hipDeviceSynchronize();

  error = hipGetLastError();
  
  if(error){
    fprintf(stderr, "Kernel launch returned %d %s\n", 
      error, hipGetErrorString(error));
    return 1;
  } else {
    fprintf(stderr, "Kernel launch successful.\n");
  }
}

